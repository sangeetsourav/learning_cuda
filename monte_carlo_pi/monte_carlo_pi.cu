// This program estimates the value of pi through Monte-Carlo simulations. It will generate points in a 1x1 square
// with a corner at (0,0) and then count how many points fall within the quadrant centered at 0,0.
// 4 * (pts inside quadrant)/(total pts) = pi

#include <hip/hip_runtime.h> // Runtime API, For cudamalloc stuff
	// For threadIdx stuff
#include <hiprand/hiprand_kernel.h> // device-side library
#include <chrono>
#include <cmath>
#include <iostream>

typedef unsigned long long int big_counter;

const int WARP_SIZE = 32;
const int WARPS_PER_BLOCK = 2;
const int THREADS_PER_BLOCK = WARPS_PER_BLOCK * WARP_SIZE;

const int NUM_BLOCKS = 2560; // Number of CUDA cores on my GPU (RTX 1000 Ada)

const big_counter ITERATIONS_PER_THREAD = 100000;

__global__ void runMonteCarlo(big_counter* total_count)
{	
	// Shared memory for all threads in a block
	__shared__ big_counter current_block_counts[THREADS_PER_BLOCK];

	// Get thread id
	int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

	hiprandState_t rng;
	// clock64() acts as the seed	
	hiprand_init(clock64(), thread_id, 0, &rng);

	// Start Monte-Carlo for this thread
	double x, y;
	current_block_counts[threadIdx.x] = 0;
	for (size_t i = 0; i < ITERATIONS_PER_THREAD; i++)
	{
		x = hiprand_uniform(&rng);
		y = hiprand_uniform(&rng);

		if (std::sqrt(x*x + y*y) <=1 )
		{
			current_block_counts[threadIdx.x] += 1;
		}
	}
	
	// Synchronization is needed because only the threads in a warp are guaranteed to be synchronised
	if (WARPS_PER_BLOCK > 1)
	{
		__syncthreads();
	}

	total_count[blockIdx.x] = 0;

	for (size_t i = 0; i < THREADS_PER_BLOCK; i++)
	{
		total_count[blockIdx.x] += current_block_counts[i];
	}

}

int main()
{
	std::cout << "Monte Carlo Simulations for estimating Pi.\n";
	std::cout << "Running on " << NUM_BLOCKS << " cores, with 1 thread-block per core and " << THREADS_PER_BLOCK << " threads per block\n";
	std::cout << "\tTotal threads = " << NUM_BLOCKS * THREADS_PER_BLOCK << "\n";
	std::cout << "\tIterations per thread = " << ITERATIONS_PER_THREAD << "\n";
	std::cout << "\tTotal random tests = " << ITERATIONS_PER_THREAD * THREADS_PER_BLOCK * NUM_BLOCKS/1000000 << " Million\n";

	auto start = std::chrono::system_clock::now();

	// We need to allocate space for 1 counter per block. Each block's counter will store how many pts were inside the quadrant
	size_t bytes = NUM_BLOCKS * sizeof(big_counter);

	big_counter* block_counters;

	hipMallocManaged(&block_counters, bytes);

	runMonteCarlo <<<NUM_BLOCKS, THREADS_PER_BLOCK>>> (block_counters);
	hipDeviceSynchronize();

	big_counter total = 0;
	for (int i = 0; i < NUM_BLOCKS; i++) {
		total += block_counters[i];
	}

	std::cout << "\tTotal random tests falling within quadrant = " << total/1000000 << " Million\n";
	std::cout << "Estimation of Pi = " << 4 * (double)total / (ITERATIONS_PER_THREAD * THREADS_PER_BLOCK * NUM_BLOCKS) << "\n";

	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> duration_in_seconds = end - start;
	std::cout << "Computation time (s) = " << duration_in_seconds.count();
}
